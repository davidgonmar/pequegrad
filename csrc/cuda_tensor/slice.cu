#include "hip/hip_runtime.h"
#include "cuda_tensor.cuh"
#include "utils.cuh"


CudaTensor CudaTensor::slice(const slice_t &slices) const {
    shape_t new_shape;
    shape_t new_strides;
    int _offset = 0;
    for (int i = 0; i < slices.size(); i++) {
        slice_item_t item = slices[i];
        if (std::holds_alternative<std::pair<int, int>>(item)) {
            // at the moment, only positive slices are supported
            auto pair = std::get<std::pair<int, int>>(item);
            int start = pair.first;
            int end = pair.second;
            PG_CHECK_ARG(start >= 0 && end >= 0, "Only positive slices are supported, got: " + std::to_string(start) + ", " + std::to_string(end));
            PG_CHECK_ARG(start < shape[i] && end <= shape[i], "Slice out of bounds, start: " + std::to_string(start) + ", end: " + std::to_string(end) + ", shape: " + std::to_string(shape[i]));
            _offset += start * strides[i];

            new_shape.push_back(end - start);
            new_strides.push_back(strides[i]);
        } else if (std::holds_alternative<int>(item)) {
            PG_CHECK_ARG(std::get<int>(item) >= 0, "Only positive slices are supported, got: " + std::to_string(std::get<int>(item)));
            PG_CHECK_ARG(std::get<int>(item) < shape[i], "Slice out of bounds, index: " + std::to_string(std::get<int>(item)) + ", shape: " + std::to_string(shape[i]));
            _offset += std::get<int>(item) * strides[i];
            // but here, since we are doing something like [:, 1], we dont add anything to the shape
            // we also dont add anything to the strides
        }
    }

    // handle the case where we dont index over ALL dimensions
    if (slices.size() < shape.size()) {
        for (int i = slices.size(); i < shape.size(); i++) {
            new_shape.push_back(shape[i]);
        }
    }

    size_t size = std::accumulate(new_shape.begin(), new_shape.end(), 1, std::multiplies<size_t>());
    CudaTensor out(size, new_shape, new_strides, ptr, dtype, _offset);

    return out;
}


CudaTensor CudaTensor::assign(const slice_t &slices, const CudaTensor &vals) {
    // We just create a sliced view of the original memory, and then copy the vals into it
    // ez pz
    const CudaTensor _sliced = this->slice(slices);

    // broadcast the vals to the shape of the sliced array. We must first remove the dimensions that are 1 on the left
    // for example, we would be trying to bc from [1, 3, 1] to [3, 1] if we didnt remove the 1s
    axes_t squeeze_dims;
    for (int i = 0; i < vals.shape.size(); i++) {
        if (vals.shape[i] != 1) {
            break;
        } else  {
            squeeze_dims.push_back(i);
        }
    }
    const CudaTensor _vals = vals.squeeze(squeeze_dims).broadcast_to(_sliced.shape).astype(_sliced.dtype);
    dim3 block_size(DEFAULT_BLOCK_SIZE);
    dim3 grid_size(ceil(_vals.size / (float)DEFAULT_BLOCK_SIZE));
    auto &sliced_strides = cuda_unique_ptr_from_host(_sliced.shape.size(), _sliced.strides.data());
    auto &sliced_shape = cuda_unique_ptr_from_host(_sliced.shape.size(), _sliced.shape.data());
    auto &vals_shape =
      cuda_unique_ptr_from_host(_vals.shape.size(), _vals.shape.data());
    auto &vals_strides =
      cuda_unique_ptr_from_host(_vals.strides.size(), _vals.strides.data());

    // copy vals into _sliced (which is a memory view of original array)
    launch_copy_with_out_strides_kernel(
      dtype, grid_size, block_size, vals_strides.get(), vals_shape.get(),
      sliced_strides.get(), sliced_shape.get(), _vals.ndim(), _sliced.ndim(), _vals.get_base_ptr(),
      _sliced.get_base_ptr());


    PG_CUDA_KERNEL_END;


    return *this;

}