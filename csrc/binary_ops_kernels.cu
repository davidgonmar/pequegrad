#include "hip/hip_runtime.h"
#include "binary_ops_kernels.cuh"
#include "binary_ops_macro.cuh"
#include "dtype.cuh"
#include <cmath>

DEF_BIN_OP_KERNEL(add_kernel, x + y, float)
DEF_BIN_OP_KERNEL(add_kernel, x + y, double)
DEF_BIN_OP_KERNEL(add_kernel, x + y, int)
DEF_BIN_OP_KERNEL(sub_kernel, x - y, float)
DEF_BIN_OP_KERNEL(sub_kernel, x - y, double)
DEF_BIN_OP_KERNEL(sub_kernel, x - y, int)
DEF_BIN_OP_KERNEL(mult_kernel, x *y, float)
DEF_BIN_OP_KERNEL(mult_kernel, x *y, double)
DEF_BIN_OP_KERNEL(mult_kernel, x *y, int)
DEF_BIN_OP_KERNEL(div_kernel, x / y, float)
DEF_BIN_OP_KERNEL(div_kernel, x / y, double)
DEF_BIN_OP_KERNEL(div_kernel, x / y, int)
DEF_BIN_OP_KERNEL(greater_kernel, x > y, float)
DEF_BIN_OP_KERNEL(greater_kernel, x > y, double)
DEF_BIN_OP_KERNEL(greater_kernel, x > y, int)
DEF_BIN_OP_KERNEL(less_kernel, x < y, float)
DEF_BIN_OP_KERNEL(less_kernel, x < y, double)
DEF_BIN_OP_KERNEL(less_kernel, x < y, int)
DEF_BIN_OP_KERNEL(equal_kernel, x == y, float)
DEF_BIN_OP_KERNEL(equal_kernel, x == y, double)
DEF_BIN_OP_KERNEL(equal_kernel, x == y, int)
DEF_BIN_OP_KERNEL(not_equal_kernel, x != y, float)
DEF_BIN_OP_KERNEL(not_equal_kernel, x != y, double)
DEF_BIN_OP_KERNEL(not_equal_kernel, x != y, int)
DEF_BIN_OP_KERNEL(greater_equal_kernel, x >= y, float)
DEF_BIN_OP_KERNEL(greater_equal_kernel, x >= y, double)
DEF_BIN_OP_KERNEL(greater_equal_kernel, x >= y, int)
DEF_BIN_OP_KERNEL(less_equal_kernel, x <= y, float)
DEF_BIN_OP_KERNEL(less_equal_kernel, x <= y, double)
DEF_BIN_OP_KERNEL(less_equal_kernel, x <= y, int)
DEF_BIN_OP_KERNEL(element_wise_max_kernel, x > y ? x : y, float)
DEF_BIN_OP_KERNEL(element_wise_max_kernel, x > y ? x : y, double)
DEF_BIN_OP_KERNEL(element_wise_max_kernel, x > y ? x : y, int)
DEF_BIN_OP_KERNEL(pow_kernel, pow(x, y), float)
DEF_BIN_OP_KERNEL(pow_kernel, pow(x, y), double)
DEF_BIN_OP_KERNEL(pow_kernel, pow(x, y), int)

void launch_binary_kernel(BinaryKernelType kernel_type, DType dtype,
                          dim3 grid_size, dim3 block_size,
                          const size_t *lhs_strides, const size_t *rhs_strides,
                          const size_t *shape, const size_t num_dims,
                          const void *lhs, const void *rhs, void *out) {
  switch (dtype) {
  case DType::Float32:
    __launch_binary_kernel<float>(
        kernel_type, grid_size, block_size, lhs_strides, rhs_strides, shape,
        num_dims, (const float *)lhs, (const float *)rhs, (float *)out);
    break;
  case DType::Float64:
    __launch_binary_kernel<double>(
        kernel_type, grid_size, block_size, lhs_strides, rhs_strides, shape,
        num_dims, (const double *)lhs, (const double *)rhs, (double *)out);
    break;
  case DType::Int32:
    __launch_binary_kernel<int>(kernel_type, grid_size, block_size, lhs_strides,
                                rhs_strides, shape, num_dims, (const int *)lhs,
                                (const int *)rhs, (int *)out);
    break;
  }
}