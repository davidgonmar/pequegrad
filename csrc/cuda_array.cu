#include "hip/hip_runtime.h"
#include "binary_ops_kernels.cuh"
#include "cuda_array.cuh"
#include "matmul_kernels.cuh"
#include "ternary_ops_kernels.cuh"
#include "unary_ops_kernels.cuh"
#include "utils.cuh"

bool CudaArray::is_contiguous() const {
  if (strides.size() != shape.size()) {
    return false;
  }
  if (strides.size() == 0) { // scalar
    return true;
  }
  shape_t expected_strides(shape.size());
  expected_strides[shape.size() - 1] = ELEM_SIZE;
  for (int i = shape.size() - 2; i >= 0; --i) {
    expected_strides[i] = expected_strides[i + 1] * shape[i + 1];
  }
  if (expected_strides != strides) {
    return false;
  }
  return true;
}

CudaArray::CudaArray(size_t size, const shape_t &shape,
                     const shape_t &strides,
                     const std::shared_ptr<float> &ptr)
    : size(size), shape(shape), strides(strides), ptr(ptr) {}

CudaArray::CudaArray(size_t size, shape_t shape, shape_t strides)
    : size(size), shape(shape), strides(strides) {
  float *raw_ptr;
  CHECK_CUDA(hipMalloc(&raw_ptr, size * ELEM_SIZE));
  ptr = std::shared_ptr<float>(raw_ptr, [](float *p) { hipFree(p); });
}

CudaArray::CudaArray(size_t size, shape_t shape) : size(size), shape(shape) {
  strides.resize(shape.size());
  // Only calculate strides if we don't have a scalar
  if (shape.size() > 0) {
    strides[shape.size() - 1] = ELEM_SIZE;
    for (int i = shape.size() - 2; i >= 0; --i) {
      strides[i] = strides[i + 1] * shape[i + 1];
    }
  }
  float *raw_ptr;
  CHECK_CUDA(hipMalloc(&raw_ptr, size * ELEM_SIZE));
  ptr = std::shared_ptr<float>(raw_ptr, [](float *p) { hipFree(p); });
}

CudaArray CudaArray::broadcast_to(const shape_t _shape) const {
  const shape_t shape_from = this->shape;
  const shape_t shape_to = _shape;
  // determine if we can broadcast
  const int from_ndim = (const int)shape_from.size();
  const int to_ndim = (const int)shape_to.size();
  // cannot broadcast if the number of dimensions of the from array is greater
  // than the number of dimensions of the to array
  if (from_ndim > to_ndim) {
    throw std::runtime_error("got incompatible shapes, to_ndim < from_ndim: " +
                             std::to_string(to_ndim) + " < " +
                             std::to_string(from_ndim));
  }

  int new_size = 1;
  shape_t new_strides(to_ndim, 0);
  // reverse test if the dim is 1 or they are equal
  for (int i = to_ndim - 1, j = from_ndim - 1; i >= 0; --i, --j) {
    py::ssize_t dim_to = shape_to[i];
    py::ssize_t dim_from =
        (j >= 0) ? shape_from[j]
                 : -1; // -1 means we 'ran' out of dimensions for j
    if (dim_to != dim_from && dim_from != 1 && dim_from != -1) {
      // we can only 'broadcast' a dimension if dim_from == 1 or we ran out of
      // dimensions.
      throw std::runtime_error("got incompatible shapes, dim_to != dim_from: " +
                               std::to_string(dim_to) +
                               " != " + std::to_string(dim_from));
    }
    if (dim_from != 1 && dim_from != -1) {
      new_strides[i] = strides[j];
    }
    new_size *= dim_to;
  }
  CudaArray out(new_size, shape_to, new_strides);
  CHECK_CUDA(hipMemcpy(out.ptr.get(), ptr.get(), size * ELEM_SIZE,
                        hipMemcpyDeviceToDevice));
  return out;
}

CudaArray CudaArray::binop(const CudaArray &other, binary_op_kernel ker) const {
  if (shape != other.shape) {
    // try to broadcast, from smaller to larger
    if (shape.size() < other.shape.size()) {
      return broadcast_to(other.shape).binop(other, ker);
    } else if (shape.size() > other.shape.size()) {
      return binop(other.broadcast_to(shape), ker);
    } else {
      // we need to check the one with less product of shape, and try to
      // broadcast
      int64_t prod_shape = 1;
      int64_t prod_other_shape = 1;
      for (int i = 0; i < shape.size(); i++) {
        prod_shape *= shape[i];
        prod_other_shape *= other.shape[i];
      }
      if (prod_shape < prod_other_shape) {
        return broadcast_to(other.shape).binop(other, ker);
      } else {
        return binop(other.broadcast_to(shape), ker);
      }
    }
  }
  assert(shape == other.shape);
  dim3 block_size(DEFAULT_BLOCK_SIZE);
  dim3 grid_size(ceil(size / (float)DEFAULT_BLOCK_SIZE));
  // Default stride calculation
  CudaArray out(size, shape);
  int n_dims = shape.size();
  int *d_strides, *d_other_strides, *d_shape;
  CHECK_CUDA(hipMalloc(&d_strides, n_dims * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_other_strides, n_dims * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_shape, n_dims * sizeof(int)));

  int *host_strides = (int *)malloc(n_dims * sizeof(int));
  int *host_other_strides = (int *)malloc(n_dims * sizeof(int));
  int *host_shape = (int *)malloc(n_dims * sizeof(int));

  for (int i = 0; i < n_dims; i++) {
    host_strides[i] = strides[i];
    host_other_strides[i] = other.strides[i];
    host_shape[i] = shape[i];
  }

  CHECK_CUDA(hipMemcpy(d_strides, host_strides, n_dims * sizeof(int),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_other_strides, host_other_strides,
                        n_dims * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_shape, host_shape, n_dims * sizeof(int),
                        hipMemcpyHostToDevice));
  ker<<<grid_size, block_size>>>(d_strides, d_other_strides, d_shape, n_dims,
                                 ptr.get(), other.ptr.get(), out.ptr.get());
  hipDeviceSynchronize();
  CHECK_CUDA(hipGetLastError());
  return out;
}

CudaArray CudaArray::ternaryop(const CudaArray &second, const CudaArray &third,
                               ternary_op_kernel ker) const {
  if (second.shape != third.shape || shape != second.shape ||
      shape != third.shape) {
    throw std::invalid_argument(
        "broadcasting is not supported in ternary operators");
  }
  dim3 block_size(DEFAULT_BLOCK_SIZE);
  dim3 grid_size(ceil(size / (float)DEFAULT_BLOCK_SIZE));

  // Default stride calculation
  CudaArray out(size, shape);
  int n_dims = shape.size();
  int *d_first_strides, *d_second_strides, *d_third_strides, *d_shape;
  CHECK_CUDA(hipMalloc(&d_first_strides, n_dims * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_second_strides, n_dims * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_third_strides, n_dims * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_shape, n_dims * sizeof(int)));

  int *host_first_strides = (int *)malloc(n_dims * sizeof(int));
  int *host_second_strides = (int *)malloc(n_dims * sizeof(int));
  int *host_third_strides = (int *)malloc(n_dims * sizeof(int));
  int *host_shape = (int *)malloc(n_dims * sizeof(int));

  for (int i = 0; i < n_dims; i++) {
    host_first_strides[i] = strides[i];
    host_second_strides[i] = second.strides[i];
    host_third_strides[i] = third.strides[i];
    host_shape[i] = shape[i];
  }

  CHECK_CUDA(hipMemcpy(d_first_strides, host_first_strides,
                        n_dims * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_second_strides, host_second_strides,
                        n_dims * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMemcpy(d_third_strides, host_third_strides,
                        n_dims * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMemcpy(d_shape, host_shape, n_dims * sizeof(int),
                        hipMemcpyHostToDevice));
  ker<<<grid_size, block_size>>>(
      d_first_strides, d_second_strides, d_third_strides, d_shape, shape.size(),
      ptr.get(), second.ptr.get(), third.ptr.get(), out.ptr.get());
  hipDeviceSynchronize();
  CHECK_CUDA(hipGetLastError());
  return out;
}

float CudaArray::getitem(shape_t index) const {
  if (index.size() != shape.size()) {
    throw std::runtime_error("Index dimension mismatch");
  }
  // Calculate the offset for the multi-dimensional index
  size_t offset = 0;
  for (size_t i = 0; i < index.size(); i++) {
    if (index[i] < 0 || index[i] >= shape[i]) {
      throw std::runtime_error("Index out of bounds");
    }
    offset += index[i] * strides[i] / ELEM_SIZE; // since strides are in bytes,
    // we need to divide by ELEM_SIZE to get the correct offset
  }
  // Copy the requested element from device to host
  float value;
  CHECK_CUDA(hipMemcpy(&value, ptr.get() + offset, ELEM_SIZE,
                        hipMemcpyDeviceToHost));
  return value;
}

int CudaArray::ndim() const { return shape.size(); }

CudaArray CudaArray::mat_mul(const CudaArray &other) const {
  CudaArray a = this->as_contiguous();
  CudaArray b = other.as_contiguous();
  dim3 block_size = dim3(DEFAULT_BLOCK_SIZE);
  shape_t new_shape;
  size_t size1, midsize, size2;
  if (a.ndim() == 2 && b.ndim() == 2) {
    size1 = a.shape.at(0);
    midsize = a.shape.at(1);
    size2 = b.shape.at(1);
    new_shape = {size1, size2};
  } else if (a.ndim() == 1 && b.ndim() == 1) {
    new_shape = {1};
    size1 = midsize = size2 = 1;
  } else if (a.ndim() == 2 && b.ndim() == 1) {
    size1 = a.shape.at(0);
    midsize = a.shape.at(1);
    size2 = 1;
    new_shape = {size1};
  } else if (a.ndim() == 1 && b.ndim() == 2) {
    size1 = 1;
    midsize = b.shape.at(0);
    size2 = b.shape.at(1);
    new_shape = {size2};
  } else {
    std::string error_message =
        "Invalid shapes for matmul, only 1D/2D combinations, 2Dx2D and 1Dx1D "
        "tensors supported";
    throw std::runtime_error(error_message);
  }

  int newSize = size1 * size2;
  dim3 gridSize(ceil(newSize / (float)DEFAULT_BLOCK_SIZE));
  CudaArray out(newSize, new_shape);
  matmul_kernel<<<gridSize, block_size>>>(a.ptr.get(), b.ptr.get(), out.ptr.get(),
                                        size1, midsize, size2);
  hipDeviceSynchronize();
  CHECK_CUDA(hipGetLastError());

  return out;
}

CudaArray CudaArray::from_numpy(py::array_t<float> np_array) {
  py::buffer_info buffer_info = np_array.request();
  std::vector<py::ssize_t> py_strides = buffer_info.strides;
  shape_t strides(py_strides.begin(), py_strides.end());
  auto size = buffer_info.size;
  auto *ptr = static_cast<float *>(buffer_info.ptr);
  std::vector<py::ssize_t> py_shape = buffer_info.shape;
  shape_t shape(py_shape.begin(), py_shape.end());
  CudaArray arr(size, shape, strides);
  CHECK_CUDA(
      hipMemcpy(arr.ptr.get(), ptr, size * ELEM_SIZE, hipMemcpyHostToDevice));
  return arr;
}

py::array_t<float> CudaArray::to_numpy() const {
  py::array_t<float> result(shape, strides);
  CHECK_CUDA(hipMemcpy(result.mutable_data(), ptr.get(), size * ELEM_SIZE,
                        hipMemcpyDeviceToHost));
  float *host = (float *)malloc(size * ELEM_SIZE);
  if (host == nullptr) {
    throw std::runtime_error("failed to allocate host memory");
  }
  hipDeviceSynchronize();
  CHECK_CUDA(
      hipMemcpy(host, ptr.get(), size * ELEM_SIZE, hipMemcpyDeviceToHost));
  return result;
}

std::string CudaArray::to_string() const {
  std::stringstream ss;
  ss << "CudaArray(" << size << ") [";
  float *host = (float *)malloc(size * ELEM_SIZE);
  if (host == nullptr) {
    throw std::runtime_error("failed to allocate host memory");
  }
  CHECK_CUDA(
      hipMemcpy(host, ptr.get(), size * ELEM_SIZE, hipMemcpyDeviceToHost));
  for (size_t i = 0; i < size; i++) {
    ss << host[i] << " ";
  }
  free(host);
  ss << "]";
  return ss.str();
}

CudaArray::~CudaArray() {}

CudaArray::CudaArray(const CudaArray &other)
    : size(other.size), shape(other.shape), strides(other.strides),
      ptr(other.ptr) {}

CudaArray &CudaArray::operator=(const CudaArray &other) {
  if (this != &other) {
    size = other.size;
    shape = other.shape;
    strides = other.strides;
    ptr = other.ptr;
  }
  return *this;
}

CudaArray::CudaArray(CudaArray &&other)
    : size(other.size), shape(std::move(other.shape)),
      strides(std::move(other.strides)), ptr(std::move(other.ptr)) {}

CudaArray &CudaArray::operator=(CudaArray &&other) {
  if (this != &other) {
    size = other.size;
    shape = std::move(other.shape);
    strides = std::move(other.strides);
    ptr = std::move(other.ptr);
  }
  return *this;
}

CudaArray CudaArray::clone() const {
  CudaArray out(size, shape, strides);
  CHECK_CUDA(hipMemcpy(out.ptr.get(), ptr.get(), size * ELEM_SIZE,
                        hipMemcpyDeviceToDevice));
  return out;
}

CudaArray CudaArray::elwiseop(element_wise_op_kernel ker) const {
  dim3 block_size(DEFAULT_BLOCK_SIZE);
  dim3 grid_size(ceil(size / (float)DEFAULT_BLOCK_SIZE));
  int n_dims = shape.size();
  int *d_strides, *d_shape;

  CHECK_CUDA(hipMalloc(&d_strides, n_dims * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_shape, n_dims * sizeof(int)));

  int *host_strides = (int *)malloc(n_dims * sizeof(int));
  int *host_shape = (int *)malloc(n_dims * sizeof(int));

  for (int i = 0; i < n_dims; i++) {
    host_strides[i] = strides[i];
    host_shape[i] = shape[i];
  }

  CHECK_CUDA(hipMemcpy(d_strides, host_strides, n_dims * sizeof(int),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_shape, host_shape, n_dims * sizeof(int),
                        hipMemcpyHostToDevice));

  CudaArray out(size, shape);
  ker<<<grid_size, block_size>>>(d_strides, d_shape, n_dims, this->ptr.get(),
                                 out.ptr.get());

  hipDeviceSynchronize();
  CHECK_CUDA(hipGetLastError());

  return out;
}

CudaArray CudaArray::as_contiguous() const { return elwiseop(copy_kernel); }

CudaArray CudaArray::permute(shape_t axes) const {
  // TODO - check that axes is from 0 to shape.size - 1, in any order
  if (axes.size() != shape.size()) {
    throw std::runtime_error("axes must have same size as shape");
  }
  shape_t new_shape(shape.size());
  shape_t new_strides(strides.size());

  for (size_t i = 0; i < axes.size(); ++i) {
    new_shape[i] = shape[axes[i]];
    new_strides[i] = strides[axes[i]];
  }

  CudaArray out(size, new_shape, new_strides, ptr);
  return out;
}