#include "mem.hpp"

std::shared_ptr<void> allocate_cuda(const size_t nbytes) {
    void *ptr;
    hipMalloc(&ptr, nbytes);
    return std::shared_ptr<void>(ptr, [](void *p) { hipFree(p); });
}

void copy_from_cpu_to_cuda(const std::shared_ptr<void> &src, const std::shared_ptr<void> &dst, const size_t nbytes) {
    hipMemcpy(dst.get(), src.get(), nbytes, hipMemcpyHostToDevice);
}

void copy_from_cuda_to_cpu(const std::shared_ptr<void> &src, const std::shared_ptr<void> &dst, const size_t nbytes) {
    hipMemcpy(dst.get(), src.get(), nbytes, hipMemcpyDeviceToHost);
}