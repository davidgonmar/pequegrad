#include "cuda_utils.cuh"
#include "mem.hpp"

std::shared_ptr<void> allocate_cuda(const size_t nbytes) {
  void *ptr;
  CHECK_CUDA(hipMallocAsync(&ptr, nbytes, 0));
  return std::shared_ptr<void>(ptr, [](void *p) { CHECK_CUDA(hipFree(p)); });
}

void copy_from_cpu_to_cuda(const std::shared_ptr<void> &src,
                           const std::shared_ptr<void> &dst,
                           const size_t nbytes) {
  hipMemcpy(dst.get(), src.get(), nbytes, hipMemcpyHostToDevice);
  CHECK_CUDA(hipGetLastError());
}

void copy_from_cpu_to_cuda(const void *src, std::shared_ptr<void> &dst,
                           const size_t nbytes) {
  hipMemcpy(dst.get(), src, nbytes, hipMemcpyHostToDevice);
  CHECK_CUDA(hipGetLastError());
}

void copy_from_cuda_to_cpu(const std::shared_ptr<void> &src,
                           const std::shared_ptr<void> &dst,
                           const size_t nbytes) {
  hipMemcpy(dst.get(), src.get(), nbytes, hipMemcpyDeviceToHost);
  CHECK_CUDA(hipGetLastError());
}

void copy_from_cuda_to_cpu(const void *src, std::shared_ptr<void> &dst,
                           const size_t nbytes) {
  hipMemcpy(dst.get(), src, nbytes, hipMemcpyDeviceToHost);
  CHECK_CUDA(hipGetLastError());
}