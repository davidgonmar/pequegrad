#include "./binary.cuh"
#include "./unary.cuh"
#include "ad_primitives.hpp"
#include "common/view_helpers.hpp"
#include "cuda_utils.cuh"
#include "random.cuh"
#include "tensor.hpp"
#include "view_helpers.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <memory>
#include <vector>

namespace pg {

void Reshape::dispatch_cuda(const std::vector<Tensor> &inputs,
                            std::vector<Tensor> &outputs) {
  CHECK_INPUTS_LENGTH(inputs, 1);
  CHECK_OUTPUTS_LENGTH(outputs, 1);
  shape_t shape = inputs[0].shape();
  Tensor a = inputs[0];
  axes_t _new_shape = _shape_to;
  shape_t new_shape(_new_shape.size());
  size_t total_new = 1;

  int neg_pos = -1;
  for (size_t i = 0; i < _new_shape.size(); i++) {
    if (_new_shape[i] < 0) {
      PG_CHECK_ARG(
          neg_pos == -1,
          "Can only specify one unknown dimension (-1) for reshape, got ",
          neg_pos, " and ", i, " for shape ", vec_to_string(_new_shape));
      neg_pos = i;
    }
    new_shape[i] = _new_shape[i];
    total_new *= new_shape[i] == -1 ? 1 : new_shape[i];
  }

  size_t total_old =
      std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<size_t>());
  if (neg_pos != -1) {
    new_shape[neg_pos] = total_old / total_new;
    PG_CHECK_ARG(
        total_old % total_new == 0,
        "New shape is not compatible with old shape: ", vec_to_string(shape),
        " not compatible with ", vec_to_string(_new_shape));
  }
  total_new = total_old;

  if (a.is_contiguous()) {
    outputs[0].init_view(std::make_shared<View>(
        view::nocopy_reshape_nocheck(a.view(), new_shape)));
    return;
  } else {
    View cont_view = cuda::view::as_contiguous(a.view());
    outputs[0].init_view(std::make_shared<View>(
        view::nocopy_reshape_nocheck(cont_view, new_shape)));
    return;
  }
}

void AsContiguous::dispatch_cuda(const std::vector<Tensor> &inputs,
                                 std::vector<Tensor> &outputs) {
  CHECK_INPUTS_LENGTH(inputs, 1);
  CHECK_OUTPUTS_LENGTH(outputs, 1);
  outputs[0].init_view(
      std::make_shared<View>(cuda::view::as_contiguous(inputs[0].view())));
}

void AsType::dispatch_cuda(const std::vector<Tensor> &inputs,
                           std::vector<Tensor> &outputs) {
  CHECK_INPUTS_LENGTH(inputs, 1);
  CHECK_OUTPUTS_LENGTH(outputs, 1);
  const Tensor &a = inputs[0];
  outputs[0].init_view(
      std::make_shared<View>(cuda::view::astype(a.view(), _dtype_to)));
}

} // namespace pg