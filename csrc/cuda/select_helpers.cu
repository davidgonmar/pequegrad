#include "hip/hip_runtime.h"
#include "ad_primitives.hpp"
#include "cuda_tensor/cuda_utils.cuh"
#include "select_helpers.cuh"
#include "shape.hpp"
#include "utils.hpp"
#include <memory>
#include <numeric>
#include <variant>
namespace pg {
CudaSelect convert_to_slice(const select_item_t &_item,
                            std::shared_ptr<Tensor> tensor = nullptr) {
  CudaSelect item;
  if (std::holds_alternative<SelectWithSlice>(_item)) {
    item.type = CudaSelectKind::SelectWithSlice;
    const auto &sss = std::get<SelectWithSlice>(_item);
    item.start = sss.start;
    item.stop = sss.stop;
    item.step = sss.step;
  } else if (std::holds_alternative<SelectWithTensor>(_item)) {
    item.type = CudaSelectKind::SelectWithTensor;
    const auto &idxArray = std::get<SelectWithTensor>(_item);
    PG_CHECK_ARG(tensor != nullptr, "Tensor is null");
    PG_CHECK_ARG(tensor->dtype() == DType::Int32,
                 "Index tensor must be of type int32");
    PG_CHECK_ARG(tensor->ndim() == 1, "Index tensor must be 1D");
    PG_CHECK_ARG(tensor->is_contiguous(), "Index tensor must be contiguous");
    auto indices = tensor->get_casted_base_ptr<int>();
    item.indices = indices;
    item.indexSize = tensor->numel();
  } else if (std::holds_alternative<SelectWithSingleIdx>(_item)) {
    item.type = CudaSelectKind::SelectWithSingleIndex;
    const auto &singleIdx = std::get<SelectWithSingleIdx>(_item);
    item.index = singleIdx.index;
  } else if (std::holds_alternative<SelectKeepDim>(_item)) {
    item.type = CudaSelectKind::SelectKeepDim;
  }
  return item;
}

namespace cuda {
void _select_with_tensor(const Tensor &inp, Tensor &outp, select_t items,
                         std::vector<Tensor> &idxs) {

  shape_t new_shape;
  int visited_tensors = 0;
  for (int i = 0; i < items.size(); i++) {
    select_item_t item = items[i];
    if (std::holds_alternative<SelectWithSlice>(item)) {
      auto _item = std::get<SelectWithSlice>(item);
      int start = _item.start;
      int stop = _item.stop;
      int step = _item.step;
      new_shape.push_back((stop - start + step - 1) / step);
    } else if (std::holds_alternative<SelectWithSingleIdx>(item)) {
      new_shape.push_back(1);
    } else if (std::holds_alternative<SelectWithTensor>(item)) {
      auto _item = std::get<SelectWithTensor>(item);
      new_shape.push_back(idxs[visited_tensors].numel());
      visited_tensors++;
    } else if (std::holds_alternative<SelectKeepDim>(item)) {
      new_shape.push_back(inp.shape()[i]);
    }
  }

  std::unique_ptr<CudaSelect[]> cuda_select_items_u =
      std::make_unique<CudaSelect[]>(items.size());
  CudaSelect *cuda_select_items = cuda_select_items_u.get();
  visited_tensors = 0;
  for (int i = 0; i < items.size(); i++) {
    std::shared_ptr<Tensor> tensor;
    if (std::holds_alternative<SelectWithTensor>(items[i])) {
      tensor = std::make_shared<Tensor>(idxs[visited_tensors]);
      visited_tensors++;
    }
    cuda_select_items[i] = convert_to_slice(items[i], tensor);
  }
  // now copy to GPU
  auto d_items = cuda_unique_ptr_from_host(items.size(), cuda_select_items);

  int total_size = std::accumulate(new_shape.begin(), new_shape.end(), 1,
                                   std::multiplies<int>());

  auto d_shape = cuda_unique_ptr_from_host(inp.ndim(), inp.shape().data());
  outp.init_view(std::make_shared<View>(new_shape, inp.dtype(), device::CUDA));
  auto out_d_shape =
      cuda_unique_ptr_from_host(outp.ndim(), outp.shape().data());
  auto src_strides =
      cuda_unique_ptr_from_host(inp.ndim(), inp.strides().data());
  int block_size = DEFAULT_BLOCK_SIZE;
  int grid_size = ceil(total_size / (float)block_size);
  switch (inp.dtype()) {
  case DType::Float32:
    _slice_and_assign_with_array_kernel<float><<<grid_size, block_size>>>(
        (float *)inp.get_base_ptr(), (float *)outp.get_base_ptr(),
        d_shape.get(), out_d_shape.get(), src_strides.get(), inp.ndim(),
        d_items.get(), items.size(), false);
    break;
  case DType::Int32:
    _slice_and_assign_with_array_kernel<int><<<grid_size, block_size>>>(
        (int *)inp.get_base_ptr(), (int *)outp.get_base_ptr(), d_shape.get(),
        out_d_shape.get(), src_strides.get(), inp.ndim(), d_items.get(),
        items.size(), false);
    break;
  case DType::Float64:
    _slice_and_assign_with_array_kernel<double><<<grid_size, block_size>>>(
        (double *)inp.get_base_ptr(), (double *)outp.get_base_ptr(),
        d_shape.get(), out_d_shape.get(), src_strides.get(), inp.ndim(),
        d_items.get(), items.size(), false);
    break;
  }

  PG_CUDA_KERNEL_END;
}
} // namespace cuda

void Select::dispatch_cuda(const std::vector<Tensor> &inputs,
                           std::vector<Tensor> &outputs) {
  shape_t new_shape;
  strides_t new_strides;
  int _offset = 0;
  bool slice_with_array = false;
  Tensor inp = inputs[0];
  PG_CHECK_ARG(inp.ndim() == _items.size(),
               "Number of slices must match number of dimensions");
  for (int i = 0; i < _items.size(); i++) {
    select_item_t item = _items[i];
    if (std::holds_alternative<SelectWithSlice>(item)) {
      // at the moment, only positive slices are supported
      auto _item = std::get<SelectWithSlice>(item);
      int start = _item.start;
      int stop = _item.stop;
      int step = _item.step;
      PG_CHECK_ARG(start < inp.shape()[i] && stop <= inp.shape()[i],
                   "Slice out of bounds, start: " + std::to_string(start) +
                       ", end: " + std::to_string(stop) +
                       ", shape: " + std::to_string(inp.shape()[i]));
      _offset += start * inp.strides()[i];
      new_shape.push_back((stop - start + step - 1) / step);
      new_strides.push_back(inp.strides()[i] * step);
    } else if (std::holds_alternative<SelectWithSingleIdx>(item)) {
      int _item = std::get<SelectWithSingleIdx>(item).index;
      PG_CHECK_ARG(_item >= 0, "Only positive slices are supported, got: " +
                                   std::to_string(_item));
      PG_CHECK_ARG(_item < inp.shape()[i], "Slice out of bounds, index: ",
                   std::to_string(_item) +
                       ", shape: " + std::to_string(inp.shape()[i]));
      _offset += _item * inp.strides()[i];
      // but here, since we are doing something like [:, 1], we dont add
      // anything to the shape we also dont add anything to the strides
    } else if (std::holds_alternative<SelectWithTensor>(item)) {
      // this is something like [:, [1, 2, 3]], where we are indexing over the i
      // dimension with an array we cant work with memory views here, so we just
      // run through a kernel to copy the values into a new array
      slice_with_array = true;
      break;
    } else if (std::holds_alternative<SelectKeepDim>(item)) {
      new_shape.push_back(inp.shape()[i]);
      new_strides.push_back(inp.strides()[i]);
    }
  }
  if (slice_with_array) {
    std::vector<Tensor> idxs =
        std::vector<Tensor>(inputs.begin() + 1, inputs.end());
    cuda::_select_with_tensor(inp, outputs[0], _items, idxs);
    return;
  }

  outputs[0].init_view(std::make_shared<View>(
      inp.view().shared_ptr(), inp.nbytes(), new_shape, new_strides,
      (size_t)_offset, inp.dtype(), inp.device()));
}

} // namespace pg
